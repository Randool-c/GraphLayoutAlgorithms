#include<hip/hip_runtime.h>

//#include<helper_cuda.h>

#include<stdio.h>

#define BLOCK_X 4
#define BLOCK_Y 4
#define N 8


__global__ void add(double *a, double *b, double *c, double *dout){
    int i = threadIdx.x + BLOCK_X * blockIdx.x;
    int j = threadIdx.y + BLOCK_Y * blockIdx.y;
    int idx = i * N + j;
    c[idx] = a[idx] + b[idx];
    __syncthreads();
    c[0] = a[0] + b[0];
    dout[0] = 1919;
    dout[1] = 9199;
}


int main(){
    double *a, *b, *out;
    double *d_a, *d_b, *d_out;
    double *iout, *dout;

    a = (double*)malloc(sizeof(double) * N * N);
    b = (double*)malloc(sizeof(double) * N * N);
    out = (double*)malloc(sizeof(double) * N * N);
    iout = (double*)malloc(sizeof(double) * 2);
    hipMalloc((void**)&d_out, sizeof(double) * N * N);
    hipMalloc((void**)&d_a, sizeof(double) * N * N);
    hipMalloc((void**)&d_b, sizeof(double) * N * N);
    hipMalloc((void**)&dout, sizeof(double) * 2);

    for (int i = 0 ; i < N * N; ++i){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMemcpy(d_a, a, sizeof(double) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * N * N, hipMemcpyHostToDevice);

    int bx = 1 + (N - 1) / BLOCK_X;
    int by = 1 + (N - 1) / BLOCK_Y;
    dim3 dimGrid(bx, by);
    dim3 dimBlock(BLOCK_X, BLOCK_Y);

    add<<<dimGrid, dimBlock>>>(d_a, d_b, d_out, dout);

    hipMemcpy(out, d_out, sizeof(double) * N * N, hipMemcpyDeviceToHost);
    hipMemcpy(iout, dout, sizeof(double) * 2, hipMemcpyDeviceToHost);
    for (int i = 0; i < N * N; ++i){
        printf("%f ", out[i]);
    }
    printf("\n");
    printf("%f\n", iout[0]);
    printf("%f\n", iout[1]);
    return 0;
}
