#include "hip/hip_runtime.h"
#include "a.h"
#include<hip/hip_runtime.h>
#include<>
//#include<hip/hip_runtime_api.h>
#include<hip/hip_runtime.h>
#define BLOCK_X 4
#define BLOCK_Y 4
#define N 8

__global__ void add_(double *a, double *b, double *c){
    int i = threadIdx.x + BLOCK_X * blockIdx.x;
    int j = threadIdx.y + BLOCK_Y * blockIdx.y;
    int idx = i * N + j;
    c[idx] = a[idx] + b[idx];
}

ClassA::ClassA(int size) {
    n = size;
    hipMalloc((void**)&d_data, sizeof(double) * size);
}

ClassA::ClassA(ClassA &&other) {
    n = other.n;
    hipMalloc((void**)&d_data, sizeof(double) * n);
    hipMemcpy(d_data, other.d_data, sizeof(double) * n, hipMemcpyDeviceToDevice);
}

ClassA ClassA::add(ClassA &other) {
    ClassA ans(N * N);
    int bx = 1 + (N - 1) / BLOCK_X;
    int by = 1 + (N - 1) / BLOCK_Y;
    dim3 dimGrid(bx, by);
    dim3 dimBlock(BLOCK_X, BLOCK_Y);
    add_<<<dimGrid, dimBlock>>>(d_data, other.d_data, ans.d_data);
    return ans;
}

ClassA::~ClassA() {
    hipFree(d_data);
}

void ClassA::to_host() {
    data = (double*)malloc(sizeof(double) * n);
    hipMemcpy(data, d_data, sizeof(double) * n, hipMemcpyDeviceToHost);
}

ClassA random(int n){
    ClassA ans(n);
    double *x = (double*)malloc(sizeof(double) * n);
    for (int i = 0; i < n; ++i){
        x[i] = rand() % 10;
        cout << x[i] << endl;
    }
    hipMemcpy(ans.d_data, x, sizeof(double) * n, hipMemcpyHostToDevice);
    return ans;
}
